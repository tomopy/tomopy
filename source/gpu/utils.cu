#include "hip/hip_runtime.h"
//  Copyright (c) 2015, UChicago Argonne, LLC. All rights reserved.
//  Copyright 2015. UChicago Argonne, LLC. This software was produced
//  under U.S. Government contract DE-AC02-06CH11357 for Argonne National
//  Laboratory (ANL), which is operated by UChicago Argonne, LLC for the
//  U.S. Department of Energy. The U.S. Government has rights to use,
//  reproduce, and distribute this software.  NEITHER THE GOVERNMENT NOR
//  UChicago Argonne, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR
//  ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.  If software is
//  modified to produce derivative works, such modified software should
//  be clearly marked, so as not to confuse it with the version available
//  from ANL.
//  Additionally, redistribution and use in source and binary forms, with
//  or without modification, are permitted provided that the following
//  conditions are met:
//      * Redistributions of source code must retain the above copyright
//        notice, this list of conditions and the following disclaimer.
//      * Redistributions in binary form must reproduce the above copyright
//        notice, this list of conditions and the following disclaimer in
//        the documentation andwith the
//        distribution.
//      * Neither the name of UChicago Argonne, LLC, Argonne National
//        Laboratory, ANL, the U.S. Government, nor the names of its
//        contributors may be used to endorse or promote products derived
//        from this software without specific prior written permission.
//  THIS SOFTWARE IS PROVIDED BY UChicago Argonne, LLC AND CONTRIBUTORS
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
//  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL UChicago
//  Argonne, LLC OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
//  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
//  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
//  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//  ---------------------------------------------------------------
//   TOMOPY CUDA implementation

//======================================================================================//

#include "common.hh"
#include "data.hh"
#include "utils.hh"

//======================================================================================//

#if defined(TOMOPY_USE_NVTX)
extern nvtxEventAttributes_t nvtx_total;
extern nvtxEventAttributes_t nvtx_iteration;
extern nvtxEventAttributes_t nvtx_slice;
extern nvtxEventAttributes_t nvtx_projection;
extern nvtxEventAttributes_t nvtx_update;
extern nvtxEventAttributes_t nvtx_rotate;
#endif

//======================================================================================//

//  gridDim:    This variable contains the dimensions of the grid.
//  blockIdx:   This variable contains the block index within the grid.
//  blockDim:   This variable and contains the dimensions of the block.
//  threadIdx:  This variable contains the thread index within the block.

//======================================================================================//
//
//  compute sum_dist
//
//======================================================================================//

__global__ void
cuda_sum_dist_compute(int dy, int dx, int nx, int ny, const int32_t* ones,
                      uint32_t* sum_dist, int p)
{
    int nx0      = blockIdx.x * blockDim.x + threadIdx.x;
    int nxstride = blockDim.x * gridDim.x;
    int dx0      = blockIdx.y * blockDim.y + threadIdx.y;
    int dxstride = blockDim.y * gridDim.y;
    int dy0      = blockIdx.z * blockDim.z + threadIdx.z;
    int dystride = blockDim.z * gridDim.z;

    for(int s = dy0; s < dy; s += dystride)
    {
        for(int d = dx0; d < dx; d += dxstride)
        {
            uint32_t*      _sum_dist = sum_dist + (s * nx * ny) + (d * nx);
            const int32_t* _ones     = ones + (d * nx);
            for(int n = nx0; n < nx; n += nxstride)
            {
                atomicAdd(&_sum_dist[n], (_ones[n] > 0) ? 1 : 0);
            }
        }
    }
}

//======================================================================================//

uint32_t*
cuda_compute_sum_dist(int dy, int dt, int dx, int nx, int ny, const float* theta)
{
    // due to some really strange issue with streams, we use the default stream here
    // because after this has been executed more than once (i.e. we do SIRT and then
    // MLEM or MLEM and then SIRT), NPP returns error code -1000.
    // it has nothing to do with algorithm strangely... and only occurs here
    // where we rotate integers. This does not affect floats...

    auto block = GetBlockDims(dim3(32, 32, 1));
    auto grid  = ComputeGridDims(dim3(nx, dt, dy), block);

    int32_t*  rot      = gpu_malloc<int32_t>(nx * ny);
    int32_t*  tmp      = gpu_malloc_and_memset<int32_t>(nx * ny, 1, 0);
    uint32_t* sum_dist = gpu_malloc_and_memset<uint32_t>(dy * nx * ny, 0, 0);
    CUDA_CHECK_LAST_ERROR();

    assert(rot != nullptr);
    assert(tmp != nullptr);
    assert(sum_dist != nullptr);

    std::this_thread::sleep_for(std::chrono::seconds(5));

    for(int p = 0; p < dt; ++p)
    {
        float theta_p_rad = fmodf(theta[p] + halfpi, twopi);
        float theta_p_deg = theta_p_rad * degrees;

        gpu_memset<int32_t>(rot, 0, nx * nx, 0);
        CUDA_CHECK_LAST_ERROR();  // debug mode only

        cuda_rotate_ip(rot, tmp, -theta_p_rad, -theta_p_deg, nx, ny, 0, GPU_NN);
        CUDA_CHECK_LAST_ERROR();  // debug mode only

        cuda_sum_dist_compute<<<grid, block, 0, 0>>>(dy, dx, nx, ny, rot, sum_dist, p);
        CUDA_CHECK_LAST_ERROR();  // debug mode only

        stream_sync(0);
    }

    // destroy
    hipFree(tmp);
    hipFree(rot);

    return sum_dist;
}

//======================================================================================//
//
//  rotate
//
//======================================================================================//

template <typename _Tp>
void
print_array(const _Tp* data, int nx, int ny, const std::string& desc)
{
    std::stringstream ss;
    ss << desc << "\n\n";
    ss << std::fixed;
    ss.precision(3);
    for(int j = 0; j < ny; ++j)
    {
        ss << "  ";
        for(int i = 0; i < nx; ++i)
        {
            ss << std::setw(8) << data[j * nx + i] << " ";
        }
        ss << std::endl;
    }
    std::cout << ss.str() << std::endl;
}

//======================================================================================//
//
//  rotate
//
//======================================================================================//

void
cuda_rotate_kernel(int32_t* dst, const int32_t* src, const float theta_rad,
                   const float theta_deg, const int nx, const int ny,
                   int eInterp = GPU_NN, hipStream_t stream = 0)
{
    stream_sync(stream);
    nppSetStream(stream);
    NVTX_RANGE_PUSH(&nvtx_rotate);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);

    auto getRotationMatrix2D = [&](double m[2][3], double scale) {
        double alpha    = scale * cos(theta_rad);
        double beta     = scale * sin(theta_rad);
        double center_x = (0.5 * nx) - 0.5;
        double center_y = (0.5 * ny) - 0.5;

        m[0][0] = alpha;
        m[0][1] = beta;
        m[0][2] = (1.0 - alpha) * center_x - beta * center_y;
        m[1][0] = -beta;
        m[1][1] = alpha;
        m[1][2] = beta * center_x + (1.0 - alpha) * center_y;
    };

    NppiSize siz;
    siz.width  = nx;
    siz.height = ny;

    NppiRect roi;
    roi.x      = 0;
    roi.y      = 0;
    roi.width  = nx;
    roi.height = ny;

    int    step = nx * sizeof(int32_t);
    double rot[2][3];
    getRotationMatrix2D(rot, 1.0);

    NppStatus ret =
        nppiWarpAffine_32s_C1R(src, siz, step, roi, dst, step, roi, rot, eInterp);

    CUDA_CHECK_LAST_STREAM_ERROR(stream);

    if(ret != NPP_SUCCESS)
        fprintf(stderr, "[%lu] %s returned non-zero NPP status: %i @ %s:%i. src = %p\n",
                GetThisThreadID(), __FUNCTION__, ret, __FILE__, __LINE__, (void*) src);

    NVTX_RANGE_POP(stream);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);
}

//======================================================================================//
//
//  rotate
//
//======================================================================================//

void
cuda_rotate_kernel(float* dst, const float* src, const float theta_rad,
                   const float theta_deg, const int nx, const int ny,
                   int eInterp = GPU_CUBIC, hipStream_t stream = 0)
{
    stream_sync(stream);
    nppSetStream(stream);
    NVTX_RANGE_PUSH(&nvtx_rotate);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);

    auto getRotationMatrix2D = [&](double m[2][3], double scale) {
        double alpha    = scale * cos(theta_rad);
        double beta     = scale * sin(theta_rad);
        double center_x = (0.5 * nx) - 0.5;
        double center_y = (0.5 * ny) - 0.5;

        m[0][0] = alpha;
        m[0][1] = beta;
        m[0][2] = (1.0 - alpha) * center_x - beta * center_y;
        m[1][0] = -beta;
        m[1][1] = alpha;
        m[1][2] = beta * center_x + (1.0 - alpha) * center_y;
    };

    NppiSize siz;
    siz.width  = nx;
    siz.height = ny;

    NppiRect roi;
    roi.x      = 0;
    roi.y      = 0;
    roi.width  = nx;
    roi.height = ny;

    int    step = nx * sizeof(float);
    double rot[2][3];
    getRotationMatrix2D(rot, 1.0);

#define USE_NPPI_ROTATE
#if defined(USE_NPPI_ROTATE)
    NppStatus ret = nppiRotate_32f_C1R(src, siz, step, roi, dst, step, roi, theta_deg,
                                       rot[0][2], rot[1][2], eInterp);
#else
    NppStatus ret =
        nppiWarpAffine_32f_C1R(src, siz, step, roi, dst, step, roi, rot, eInterp);
#endif

    CUDA_CHECK_LAST_STREAM_ERROR(stream);

    if(ret != NPP_SUCCESS)
        fprintf(stderr, "[%lu] %s returned non-zero NPP status: %i @ %s:%i. src = %p\n",
                GetThisThreadID(), __FUNCTION__, ret, __FILE__, __LINE__, (void*) src);

    NVTX_RANGE_POP(stream);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);
}

//======================================================================================//

__global__ void
cuda_rotate_internal_kernel(float* dst, const float* src, float theta, const int nx,
                            const int ny)
{
    // this is flawed and should not be production
    int   src_size = nx * ny;
    float xoff     = (0.5f * nx) - 0.5f;
    float yoff     = (0.5f * ny) - 0.5f;

    int j0      = blockIdx.x * blockDim.x + threadIdx.x;
    int jstride = blockDim.x * gridDim.x;

    for(int j = j0; j < ny; j += jstride)
    {
        for(int i = 0; i < nx; ++i)
        {
            // indices in 2D
            float rx = float(i) - xoff;
            float ry = float(j) - yoff;
            // transformation
            float tx = rx * cosf(theta) + -ry * sinf(theta);
            float ty = rx * sinf(theta) + ry * cosf(theta);
            // indices in 2D
            float x = (tx + xoff);
            float y = (ty + yoff);
            // index in 1D array
            int  rz    = j * nx + i;
            auto index = [&](int _x, int _y) { return _y * nx + _x; };
            // within bounds
            int   x1    = floorf(tx + xoff);
            int   y1    = floorf(ty + yoff);
            int   x2    = x1 + 1;
            int   y2    = y1 + 1;
            float fxy1  = 0.0f;
            float fxy2  = 0.0f;
            int   ixy11 = index(x1, y1);
            int   ixy21 = index(x2, y1);
            int   ixy12 = index(x1, y2);
            int   ixy22 = index(x2, y2);
            if(ixy11 >= 0 && ixy11 < src_size)
                fxy1 += (x2 - x) * src[ixy11];
            if(ixy21 >= 0 && ixy21 < src_size)
                fxy1 += (x - x1) * src[ixy21];
            if(ixy12 >= 0 && ixy12 < src_size)
                fxy2 += (x2 - x) * src[ixy12];
            if(ixy22 >= 0 && ixy22 < src_size)
                fxy2 += (x - x1) * src[ixy22];
            dst[rz] += (y2 - y) * fxy1 + (y - y1) * fxy2;
        }
    }
}

//======================================================================================//

int32_t*
cuda_rotate(const int32_t* src, const float theta_rad, const float theta_deg,
            const int nx, const int ny, hipStream_t stream, const int eInterp)
{
    int32_t* _dst = gpu_malloc<int32_t>(nx * ny);
    cuda_rotate_kernel(_dst, src, theta_rad, theta_deg, nx, ny, eInterp, stream);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);
    return _dst;
}

//======================================================================================//

void
cuda_rotate_ip(int32_t* dst, const int32_t* src, const float theta_rad,
               const float theta_deg, const int nx, const int ny, hipStream_t stream,
               const int eInterp)
{
    cuda_rotate_kernel(dst, src, theta_rad, theta_deg, nx, ny, eInterp, stream);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);
}

//======================================================================================//

float*
cuda_rotate(const float* src, const float theta_rad, const float theta_deg, const int nx,
            const int ny, hipStream_t stream, const int eInterp)
{
    float* _dst = gpu_malloc<float>(nx * ny);
    cuda_rotate_kernel(_dst, src, theta_rad, theta_deg, nx, ny, eInterp, stream);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);
    return _dst;
}

//======================================================================================//

void
cuda_rotate_ip(float* dst, const float* src, const float theta_rad, const float theta_deg,
               const int nx, const int ny, hipStream_t stream, const int eInterp)
{
    cuda_rotate_kernel(dst, src, theta_rad, theta_deg, nx, ny, eInterp, stream);
    CUDA_CHECK_LAST_STREAM_ERROR(stream);
}

//======================================================================================//
