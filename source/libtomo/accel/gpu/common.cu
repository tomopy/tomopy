#include "hip/hip_runtime.h"
//  Copyright (c) 2015, UChicago Argonne, LLC. All rights reserved.
//  Copyright 2015. UChicago Argonne, LLC. This software was produced
//  under U.S. Government contract DE-AC02-06CH11357 for Argonne National
//  Laboratory (ANL), which is operated by UChicago Argonne, LLC for the
//  U.S. Department of Energy. The U.S. Government has rights to use,
//  reproduce, and distribute this software.  NEITHER THE GOVERNMENT NOR
//  UChicago Argonne, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR
//  ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.  If software is
//  modified to produce derivative works, such modified software should
//  be clearly marked, so as not to confuse it with the version available
//  from ANL.
//  Additionally, redistribution and use in source and binary forms, with
//  or without modification, are permitted provided that the following
//  conditions are met:
//      * Redistributions of source code must retain the above copyright
//        notice, this list of conditions and the following disclaimer.
//      * Redistributions in binary form must reproduce the above copyright
//        notice, this list of conditions and the following disclaimer in
//        the documentation andwith the
//        distribution.
//      * Neither the name of UChicago Argonne, LLC, Argonne National
//        Laboratory, ANL, the U.S. Government, nor the names of its
//        contributors may be used to endorse or promote products derived
//        from this software without specific prior written permission.
//  THIS SOFTWARE IS PROVIDED BY UChicago Argonne, LLC AND CONTRIBUTORS
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
//  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL UChicago
//  Argonne, LLC OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
//  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
//  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
//  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//  ---------------------------------------------------------------
//   TOMOPY CUDA implementation

#include "common.hh"
#include "macros.hh"
#include "utils.hh"
#include "hip/hip_runtime.h"

//======================================================================================//

#if defined(TOMOPY_USE_NVTX)

nvtxEventAttributes_t nvtx_total;
nvtxEventAttributes_t nvtx_iteration;
nvtxEventAttributes_t nvtx_slice;
nvtxEventAttributes_t nvtx_projection;
nvtxEventAttributes_t nvtx_update;
nvtxEventAttributes_t nvtx_rotate;

//--------------------------------------------------------------------------------------//

void
init_nvtx()
{
    static bool first = true;
    if(!first)
        return;
    first = false;

    nvtx_total.version       = NVTX_VERSION;
    nvtx_total.size          = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    nvtx_total.colorType     = NVTX_COLOR_ARGB;
    nvtx_total.color         = 0xff0000ff; /* blue? */
    nvtx_total.messageType   = NVTX_MESSAGE_TYPE_ASCII;
    nvtx_total.message.ascii = "total time for all iterations";

    nvtx_iteration.version       = NVTX_VERSION;
    nvtx_iteration.size          = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    nvtx_iteration.colorType     = NVTX_COLOR_ARGB;
    nvtx_iteration.color         = 0xffffff00; /* yellow */
    nvtx_iteration.messageType   = NVTX_MESSAGE_TYPE_ASCII;
    nvtx_iteration.message.ascii = "time per iteration";

    nvtx_slice.version       = NVTX_VERSION;
    nvtx_slice.size          = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    nvtx_slice.colorType     = NVTX_COLOR_ARGB;
    nvtx_slice.color         = 0xff00ffff; /* cyan */
    nvtx_slice.messageType   = NVTX_MESSAGE_TYPE_ASCII;
    nvtx_slice.message.ascii = "time per slice";

    nvtx_projection.version       = NVTX_VERSION;
    nvtx_projection.size          = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    nvtx_projection.colorType     = NVTX_COLOR_ARGB;
    nvtx_projection.color         = 0xff00ffff; /* pink */
    nvtx_projection.messageType   = NVTX_MESSAGE_TYPE_ASCII;
    nvtx_projection.message.ascii = "time per projection";

    nvtx_update.version       = NVTX_VERSION;
    nvtx_update.size          = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    nvtx_update.colorType     = NVTX_COLOR_ARGB;
    nvtx_update.color         = 0xff99ff99; /* light green */
    nvtx_update.messageType   = NVTX_MESSAGE_TYPE_ASCII;
    nvtx_update.message.ascii = "time updating";

    nvtx_rotate.version       = NVTX_VERSION;
    nvtx_rotate.size          = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    nvtx_rotate.colorType     = NVTX_COLOR_ARGB;
    nvtx_rotate.color         = 0xff0000ff; /* blue? */
    nvtx_rotate.messageType   = NVTX_MESSAGE_TYPE_ASCII;
    nvtx_rotate.message.ascii = "time rotating";
}

#endif

//======================================================================================//

int
cuda_set_device(int device)
{
    int deviceCount = cuda_device_count();
    if(deviceCount == 0)
        return -1;

    // don't set to higher than number of devices
    device = device % deviceCount;
    // update thread-static variable
    this_thread_device() = device;
    // actually set the device
    hipSetDevice(device);
    // return the modulus
    return device;
}

//======================================================================================//

int
cuda_multi_processor_count()
{
    if(cuda_device_count() == 0)
        return 0;

    // keep from querying device
    static thread_local cuda_device_info<int>* _instance = new cuda_device_info<int>();
    // use the thread assigned devices
    int device = this_thread_device();

    if(_instance->find(device) != _instance->end())
        return _instance->find(device)->second;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    return ((*_instance)[device] = deviceProp.multiProcessorCount);
}

//======================================================================================//

int
cuda_max_threads_per_block()
{
    if(cuda_device_count() == 0)
        return 0;

    // keep from querying device
    static thread_local cuda_device_info<int>* _instance = new cuda_device_info<int>();
    // use the thread assigned devices
    int device = this_thread_device();

    if(_instance->find(device) != _instance->end())
        return _instance->find(device)->second;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    return ((*_instance)[device] = deviceProp.maxThreadsPerBlock);
}

//======================================================================================//

int
cuda_warp_size()
{
    if(cuda_device_count() == 0)
        return 0;

    // keep from querying device
    static thread_local cuda_device_info<int>* _instance = new cuda_device_info<int>();
    // use the thread assigned devices
    int device = this_thread_device();

    if(_instance->find(device) != _instance->end())
        return _instance->find(device)->second;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    return ((*_instance)[device] = deviceProp.warpSize);
}

//======================================================================================//

int
cuda_shared_memory_per_block()
{
    if(cuda_device_count() == 0)
        return 0;

    // keep from querying device
    static thread_local cuda_device_info<int>* _instance = new cuda_device_info<int>();
    // use the thread assigned devices
    int device = this_thread_device();

    if(_instance->find(device) != _instance->end())
        return _instance->find(device)->second;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    return ((*_instance)[device] = deviceProp.sharedMemPerBlock);
}

//======================================================================================//

int
cuda_device_count()
{
    int         deviceCount = 0;
    hipError_t error_id    = hipGetDeviceCount(&deviceCount);

    if(error_id != hipSuccess)
        return 0;

    return deviceCount;
}

//======================================================================================//

void
cuda_device_query()
{
    auto pythreads = GetEnv("TOMOPY_PYTHON_THREADS", HW_CONCURRENCY);
    static std::atomic<int16_t> _once;
    auto                        _count = _once++;
    if(_count + 1 == pythreads)
        _once.store(0);
    if(_count > 0)
        return;

    int         deviceCount    = 0;
    int         driverVersion  = 0;
    int         runtimeVersion = 0;
    hipError_t error_id       = hipGetDeviceCount(&deviceCount);

    if(error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned error code %d\n--> %s\n",
               static_cast<int>(error_id), hipGetErrorString(error_id));

        if(deviceCount > 0)
        {
            hipSetDevice(0);
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, 0);
            printf("\nDevice %d: \"%s\"\n", 0, deviceProp.name);

            // Console log
            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);
            printf("  CUDA Driver Version / Runtime Version          %d.%d / "
                   "%d.%d\n",
                   driverVersion / 1000, (driverVersion % 100) / 10,
                   runtimeVersion / 1000, (runtimeVersion % 100) / 10);
            printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
                   deviceProp.major, deviceProp.minor);
        }

        return;
    }

    AutoLock l(TypeMutex<decltype(std::cout)>());

    if(deviceCount == 0)
        printf("No available CUDA device(s) detected\n");
    else
        printf("Detected %d CUDA capable devices\n", deviceCount);

    for(int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        // Console log
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);

        printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
               driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000,
               (runtimeVersion % 100) / 10);

        printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
               deviceProp.major, deviceProp.minor);

        char msg[256];
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
        sprintf_s(msg, sizeof(msg),
                  "  Total amount of global memory:                 %.0f MBytes "
                  "(%llu bytes)\n",
                  static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
                  (unsigned long long) deviceProp.totalGlobalMem);
#else
        snprintf(msg, sizeof(msg),
                 "  Total amount of global memory:                 %.0f MBytes "
                 "(%llu bytes)\n",
                 static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
                 (unsigned long long) deviceProp.totalGlobalMem);
#endif
        printf("%s", msg);

        printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f "
               "GHz)\n",
               deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

#if CUDART_VERSION >= 5000
        // This is supported in CUDA 5.0 (runtime API device properties)
        printf("  Memory Clock rate:                             %.0f Mhz\n",
               deviceProp.memoryClockRate * 1e-3f);
        printf("  Memory Bus Width:                              %d-bit\n",
               deviceProp.memoryBusWidth);

        if(deviceProp.l2CacheSize)
        {
            printf("  L2 Cache Size:                                 %d bytes\n",
                   deviceProp.l2CacheSize);
        }

#else
        // This only available in CUDA 4.0-4.2 (but these were only exposed in
        // the CUDA Driver API)
        int memoryClock;
        int memBusWidth;
        int L2CacheSize;
        getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
        printf("  Memory Clock rate:                             %.0f Mhz\n",
               memoryClock * 1e-3f);
        getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth,
                              dev);
        printf("  Memory Bus Width:                              %d-bit\n", memBusWidth);
        getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

        if(L2CacheSize)
            printf("  L2 Cache Size:                                 %d bytes\n",
                   L2CacheSize);
#endif

        printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, "
               "%d), 3D=(%d, %d, %d)\n",
               deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
               deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
               deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
        printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d "
               "layers\n",
               deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
        printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d "
               "layers\n",
               deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
               deviceProp.maxTexture2DLayered[2]);

        printf("  Total amount of constant memory:               %lu bytes\n",
               deviceProp.totalConstMem);
        printf("  Total amount of shared memory per block:       %lu bytes\n",
               deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n",
               deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n",
               deviceProp.warpSize);
        printf("  Multiprocessor count:                          %d\n",
               deviceProp.multiProcessorCount);
        printf("  Maximum number of threads per multiprocessor:  %d\n",
               deviceProp.maxThreadsPerMultiProcessor);
        printf("  Maximum number of threads per block:           %d\n",
               deviceProp.maxThreadsPerBlock);
        printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %lu bytes\n",
               deviceProp.memPitch);
        printf("  Texture alignment:                             %lu bytes\n",
               deviceProp.textureAlignment);
        printf("  Concurrent copy and kernel execution:          %s with %d copy "
               "engine(s)\n",
               (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
        printf("  Run time limit on kernels:                     %s\n",
               deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
        printf("  Integrated GPU sharing Host Memory:            %s\n",
               deviceProp.integrated ? "Yes" : "No");
        printf("  Support host page-locked memory mapping:       %s\n",
               deviceProp.canMapHostMemory ? "Yes" : "No");
        printf("  Alignment requirement for Surfaces:            %s\n",
               deviceProp.surfaceAlignment ? "Yes" : "No");
        printf("  Device has ECC support:                        %s\n",
               deviceProp.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
        printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n",
               deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)"
                                    : "WDDM (Windows Display Driver Model)");
#endif
        printf("  Device supports Unified Addressing (UVA):      %s\n",
               deviceProp.unifiedAddressing ? "Yes" : "No");
        printf("  Device supports Compute Preemption:            %s\n",
               deviceProp.computePreemptionSupported ? "Yes" : "No");
        printf("  Supports Cooperative Kernel Launch:            %s\n",
               deviceProp.cooperativeLaunch ? "Yes" : "No");
        printf("  Supports MultiDevice Co-op Kernel Launch:      %s\n",
               deviceProp.cooperativeMultiDeviceLaunch ? "Yes" : "No");
        printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n",
               deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);

        const char* sComputeMode[] = {
            "Default (multiple host threads can use ::hipSetDevice() with "
            "device "
            "simultaneously)",
            "Exclusive (only one host thread in one process is able to use "
            "::hipSetDevice() with this device)",
            "Prohibited (no host thread can use ::hipSetDevice() with this "
            "device)",
            "Exclusive Process (many threads in one process is able to use "
            "::hipSetDevice() with this device)",
            "Unknown",
            nullptr
        };
        printf("  Compute Mode:\n");
        printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
    }

    printf("\n\n");
    CUDA_CHECK_LAST_ERROR();
}

//======================================================================================//
